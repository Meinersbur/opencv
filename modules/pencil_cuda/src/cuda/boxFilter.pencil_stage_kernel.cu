
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
__global__ void __ppcg_applyBoxFilter_kernel0(unsigned char *dst, unsigned char *src, int srcCols, int rows, int dstCols, int cols, int se_dim, int filterSizeY, int filterSizeX, int filterAnchorX, int filterAnchorY)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.y, t1 = threadIdx.x;
    int private_avg_sum;
    int private_curr_y;
    int private_curr_x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    for (int c0 = 32 * b0; c0 < rows; c0 += 8192)
      if (rows >= t0 + c0 + 1)
        for (int c1 = 32 * b1; c1 < cols; c1 += 8192)
          for (int c3 = t1; c3 <= ppcg_min(31, cols - c1 - 1); c3 += 16) {
            private_avg_sum = 0;
            for (int c4 = 0; c4 < filterSizeY; c4 += 1) {
              private_curr_y = (-filterAnchorY + t0 + c0 + c4);
              if ((-filterAnchorY + t0 + c0 + c4) < 0) {
                private_curr_y = (-(-filterAnchorY + t0 + c0 + c4));
              } else {
                if ((-filterAnchorY + t0 + c0 + c4) >= (rows)) {
                  private_curr_y = ((2 * ((rows) - 1)) - (-filterAnchorY + t0 + c0 + c4));
                }
              }
              for (int c5 = 0; c5 < filterSizeX; c5 += 1) {
                private_curr_x = (-filterAnchorX + c1 + c3 + c5);
                if ((-filterAnchorX + c1 + c3 + c5) < 0) {
                  private_curr_x = (-(-filterAnchorX + c1 + c3 + c5));
                } else {
                  if ((-filterAnchorX + c1 + c3 + c5) >= (cols)) {
                    private_curr_x = ((2 * ((cols) - 1)) - (-filterAnchorX + c1 + c3 + c5));
                  }
                }
                private_avg_sum += src[private_curr_y * srcCols + private_curr_x];
              }
            }
            dst[(t0 + c0) * dstCols + (c1 + c3)] = (private_avg_sum / (se_dim));
          }
}

extern "C" void __ppcg_applyBoxFilter_prog0(int srcCols, int rows, int dstCols, int cols, int se_dim, int filterSizeY, int filterSizeX, int filterAnchorX, int filterAnchorY, unsigned char *dst, unsigned char *src)
{
  #define ppcg_fdiv_q(n,d) (((n)<0) ? -((-(n)+(d)-1)/(d)) : (n)/(d))
  #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
  {
#define cudaCheckReturn(ret) \
  do { \
    hipError_t cudaCheckReturn_e = (ret); \
    if (cudaCheckReturn_e != hipSuccess) { \
      fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaCheckReturn_e)); \
      fflush(stderr); \
    } \
    assert(cudaCheckReturn_e == hipSuccess); \
  } while(0)
#define cudaCheckKernel() \
  do { \
    cudaCheckReturn(hipGetLastError()); \
  } while(0)
/*
    unsigned char *dev_dst;
    unsigned char *dev_src;
    
    cudaCheckReturn(cudaMalloc((void **) &dev_dst, (rows) * (dstCols) * sizeof(unsigned char)));
    cudaCheckReturn(cudaMalloc((void **) &dev_src, (rows) * (srcCols) * sizeof(unsigned char)));
    
    cudaCheckReturn(cudaMemcpy(dev_dst, dst, (rows) * (dstCols) * sizeof(unsigned char), cudaMemcpyHostToDevice));
    cudaCheckReturn(cudaMemcpy(dev_src, src, (rows) * (srcCols) * sizeof(unsigned char), cudaMemcpyHostToDevice));*/
    {
      dim3 k0_dimBlock(16, 32);
      dim3 k0_dimGrid(ppcg_min((cols + 31) / 32, 256), ppcg_min(256, (rows + 31) / 32));
      __ppcg_applyBoxFilter_kernel0 <<<k0_dimGrid, k0_dimBlock>>> (dst, src, srcCols, rows, dstCols, cols, se_dim, filterSizeY, filterSizeX, filterAnchorX, filterAnchorY);
      cudaCheckKernel();
    }
    /*
    cudaCheckReturn(cudaMemcpy(dst, dev_dst, (rows) * (dstCols) * sizeof(unsigned char), cudaMemcpyDeviceToHost));
    cudaCheckReturn(cudaFree(dev_dst));
    cudaCheckReturn(cudaFree(dev_src));*/
  }
}
